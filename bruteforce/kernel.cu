#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <string>
#include <time.h>
#include <stdlib.h>
#include <windows.h>
#include "hip/hip_runtime.h"

#define HASH_LEN 3
//#define BENCHMARK
#define ARG_COUNT 4
//#define ALPHABET_COUNT 75
#define ALPHABET_COUNT 75
#define ALPHABET_START 48

using namespace std;

hipError_t bruteForceWithCuda(char* hostHash, int blockCount, int threadCount);

__device__ char* devHash;
__device__ bool* stopGlobal;


static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}


__device__ bool my_strcpm(const char *string1, const char *string2, int size)
{
	for (int i = 0; i < size; i++)
	{
		if (string1[i] != string2[i])
			return false;
	}
	return true;
}

__device__ char * my_strcpy(char *dest, const char *src) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
	return dest;
}

__device__ char* rot(char* rotatedPassword, int rotBase, int* devHashLength)
{
	for (int i = 0; i < *devHashLength; i++)
	{
		rotatedPassword[i] = ((rotatedPassword[i] - ALPHABET_START + rotBase) % ALPHABET_COUNT) + ALPHABET_START;
	}
	return rotatedPassword;
}

__device__ char* offsetToWord(int offset, int hashSize)
{
	char sampleWord[HASH_LEN + 1];
	if (sampleWord == NULL) {
		printf("null pointer sampleword\n");
	}
	for (int i = 0; i < hashSize; i++)
	{
		sampleWord[i] = ALPHABET_START;
	}
	sampleWord[hashSize] = '\0';
	int i = hashSize - 1;
	while (offset > 0)
	{
		int x = offset % ALPHABET_COUNT;
		sampleWord[i] = ALPHABET_START + x;
		offset = offset / ALPHABET_COUNT;
		i--;
	}
	return sampleWord;
}

__global__ void addKernel(char* hashPwd, int* hashLength, bool* stopGlobal)
{
	__shared__ bool stop[1];
	stop[0] = false;

	__syncthreads();
	clock_t start = clock();
	unsigned int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
	int threadCount = blockDim.x * blockDim.y * blockDim.z * gridDim.x * gridDim.y * gridDim.z;
	int combinationCount = pow((double)ALPHABET_COUNT, (double)*hashLength);
	int combinationPerThread = ceil((double)combinationCount / (double)threadCount);
	int offset = combinationPerThread * threadId;
	char rotatedPassword[HASH_LEN + 1];
	for (int i = offset; i < offset + combinationPerThread; i++)
	{


		if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
		{
			if (*stopGlobal)
			{
				stop[0] = true;
				return;
			}
			if (stop[0])
			{
				*stopGlobal = true;
				return;
			}
		}
		else 
		{
			if (stop[0])
			{
				return;
				
			}
		}
		char* sampleWord = offsetToWord(i, *hashLength);

		my_strcpy(rotatedPassword, sampleWord);

		rot(rotatedPassword, 13, hashLength);

		if (my_strcpm(rotatedPassword, hashPwd, *hashLength))
		{
			stop[0] = true;
			clock_t stop = clock();
			printf("succes: %s  offset: %d offsetWord: %s\n", rotatedPassword, offset, sampleWord);
			
		}
	}
}


int main(int argc, char* argv[])
{
	if (argc != ARG_COUNT) {
		printf("Usage ./cudaexample blockCount threadCount hashPassowrd");
		exit(1);
	}

	if (strlen(argv[ARG_COUNT - 1]) > 7) {
		printf("Too big hash size");
		exit(1);
	}
	char* hostHash = (char*)malloc(strlen(argv[ARG_COUNT-1]));
	HANDLE_NULL(hostHash);
	strcpy(hostHash, argv[ARG_COUNT - 1]);

	int blockCount = atoi(argv[1]);
	int threadCount = atoi(argv[2]);
	hipError_t cudaStatus = bruteForceWithCuda(hostHash, blockCount, threadCount);

    return 0;
}

hipError_t bruteForceWithCuda(char* hostHash, int blockCount, int threadCount)
{
	hipError_t cudaStatus;
	for (int block_count = 1; block_count < 2048; block_count *= 2)
	{
	cudaStatus = hipSetDevice(0);
	HANDLE_ERROR(cudaStatus);
	
	int* hostHashLength = (int*)malloc(sizeof(int));
	HANDLE_NULL(hostHashLength);
	*hostHashLength = strlen(hostHash);
	int* devHashLength;

	HANDLE_ERROR(hipMalloc((void**)&devHashLength, sizeof(int)));

	HANDLE_ERROR(hipMemcpy(devHashLength, hostHashLength, sizeof(int), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMalloc((void**)&devHash, strlen(hostHash)));

	HANDLE_ERROR(hipMemcpy(devHash, hostHash, strlen(hostHash), hipMemcpyHostToDevice));

	bool* stop = (bool*)malloc(sizeof(bool));
	HANDLE_NULL(stop);
	*stop = false;

	HANDLE_ERROR(hipMalloc((void**)&stopGlobal, sizeof(bool)));
	HANDLE_ERROR(hipMemcpy(stopGlobal, stop, sizeof(bool), hipMemcpyHostToDevice));

	hipDeviceProp_t deviceProperties;
	hipGetDeviceProperties(&deviceProperties, 0);
	int max_threads;
	int max_blocks;
	max_threads = deviceProperties.maxThreadsPerBlock;


	hipEvent_t start;
	hipEvent_t stop1;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop1));
	HANDLE_ERROR(hipEventRecord(start, 0));

	addKernel <<<1, block_count>>> (devHash, devHashLength, stopGlobal);

	
	HANDLE_ERROR(hipGetLastError());

	//HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipEventRecord(stop1, 0));
	HANDLE_ERROR(hipEventSynchronize(stop1));
	float   elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
		start, stop1));
	printf("Czas generowania:  %3.1f ms ilosc blokow: %d\n", elapsedTime, block_count);

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop1));

	}

	return cudaStatus;
}